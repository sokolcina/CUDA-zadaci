#include "hip/hip_runtime.h"
#include "../common/common.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include "../common/stopwatch.h"

#define NSTREAM 4



void initialData(float *in,  const int size)
{
    for (int i = 0; i < size; i++)
    {
        in[i] = (float)( rand() & 0xFF ) / 10.0f; //100.0f;
    }

    return;
}


__global__ void a1  (float *A,float *B,float *C, const int ncols,const int nrows)
{
    const unsigned int j=blockIdx.x*blockDim.x+threadIdx.x;
    const unsigned int i=blockIdx.y*blockDim.y+threadIdx.y;
    const unsigned int id=ncols*i+j;
    const unsigned int N = nrows * ncols;
    if(id<N)
        C[id]=A[id]+B[id];
}

__global__ void a2  (float *A,float *B,float *C, const int ncols,const int nrows)
{
    const unsigned int j=blockIdx.x*blockDim.x+threadIdx.x;
    const unsigned int i=blockIdx.y*blockDim.y+threadIdx.y;
    const unsigned int id=ncols*i+j;
    const unsigned int N = nrows * ncols;
    if(id<N)
        C[id]=A[id]+B[id];
}

__global__ void s1  (float *A,float *B,float *C, const int ncols,const int nrows)
{
    const unsigned int j=blockIdx.x*blockDim.x+threadIdx.x;
    const unsigned int i=blockIdx.y*blockDim.y+threadIdx.y;
    const unsigned int id=ncols*i+j;
    const unsigned int N = nrows * ncols;
    if(id<N)
        C[id]=A[id]-B[id];
}
__global__ void s2  (float *A,float *B,float *C, const int ncols,const int nrows)
{
    const unsigned int j=blockIdx.x*blockDim.x+threadIdx.x;
    const unsigned int i=blockIdx.y*blockDim.y+threadIdx.y;
    const unsigned int id=ncols*i+j;
    const unsigned int N = nrows * ncols;
    if(id<N)
        C[id]=A[id]-B[id];
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("%s starting main at ", argv[0]);
    printf("device %d: %s \n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));
    

   
    int nx = 1 << 12;
    int ny = 1 << 12;

   
    int blockx = 16;
    int blocky = 16;

    if (argc > 1) blockx = atoi(argv[1]);

    if (argc > 2) blocky  = atoi(argv[2]);

    if (argc > 3) nx  = atoi(argv[3]);

    if (argc > 4) ny  = atoi(argv[4]);


    size_t nBytes = nx * ny * sizeof(float);

    int N = nx * ny;
    // execution configuration
    dim3 block (blockx, blocky);
    dim3 grid  ((nx + block.x - 1) / block.x, (ny + block.y - 1) / block.y);

    // allocate host memory
    float *h_A = (float *)malloc(nBytes);
    float *h_B = (float *)malloc(nBytes);
    float *h_res1 = (float *)malloc(nBytes);
    float *h_res2  = (float *)malloc(nBytes);
    float *h_res3  = (float *)malloc(nBytes);
    float *h_res4  = (float *)malloc(nBytes);

    float *d_A1, *d_A2, *d_A3, *d_A4;
    float *d_B1, *d_B2, *d_B3, *d_B4;
    float *d_C1, *d_C2, *d_C3, *d_C4;

    CHECK(hipMalloc((void **)&d_A1, nBytes));
    CHECK(hipMalloc((void **)&d_A2, nBytes));
    CHECK(hipMalloc((void **)&d_A3, nBytes));
    CHECK(hipMalloc((void **)&d_A4, nBytes));

    CHECK(hipMalloc((void **)&d_B1, nBytes));
    CHECK(hipMalloc((void **)&d_B2, nBytes));
    CHECK(hipMalloc((void **)&d_B3, nBytes));
    CHECK(hipMalloc((void **)&d_B4, nBytes));

    CHECK(hipMalloc((void **)&d_C1, nBytes));
    CHECK(hipMalloc((void **)&d_C2, nBytes));
    CHECK(hipMalloc((void **)&d_C3, nBytes));
    CHECK(hipMalloc((void **)&d_C4, nBytes));

    initialData(h_A, N);
    initialData(h_B, N);


    CHECK(hipMemcpy(d_A1, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_A2, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_A3, h_A, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_A4, h_A, nBytes, hipMemcpyHostToDevice));

    CHECK(hipMemcpy(d_B1, h_B, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B2, h_B, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B3, h_B, nBytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_B4, h_B, nBytes, hipMemcpyHostToDevice));
    // creat events
    hipEvent_t start, stop, e_A,e_S;
    CHECK(hipEventCreate(&start));
    CHECK(hipEventCreate(&stop));

    Stopwatch s;
    // record start event
    CHECK(hipEventRecord(start, 0));

    a1<<<grid,block>>>(d_A1,d_B1,d_C1,nx,ny);
    a2<<<grid,block>>>(d_A2,d_B2,d_C2,nx,ny);
    s1<<<grid,block>>>(d_A3,d_B3,d_C3,nx,ny);
    s2<<<grid,block>>>(d_A4,d_B4,d_C4,nx,ny);

    // record stop event
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));

    float elapsed_time;
    // calculate elapsed time
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Measured time for sequence execution = %f\nstopwatch = %f\n",
           elapsed_time / 1000.0f,s.elapsed());


    CHECK(hipMemcpy(h_res1, d_C1, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_res2, d_C2, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_res3, d_C3, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_res4, d_C4, nBytes, hipMemcpyDeviceToHost));
    for(int i=0;i<N;i++)
    {
        if(h_res1[i]!=h_res2[i] || h_res3[i]!=h_res4[i])
            printf("BAD\n");
    }
    // Allocate and initialize an array of stream handles
    hipStream_t *streams = (hipStream_t *) malloc(NSTREAM * sizeof(
        hipStream_t));

    for (int i = 0 ; i < NSTREAM ; i++)
    {
        CHECK(hipStreamCreate(&(streams[i])));
    }

     // record start event
     s.reset();
     CHECK(hipEventRecord(start, 0));
   
     a1<<<grid,block,0,streams[0]>>>(d_A1,d_B1,d_C1,nx,ny);
     a2<<<grid,block,0,streams[1]>>>(d_A2,d_B2,d_C2,nx,ny);
     s1<<<grid,block,0,streams[2]>>>(d_A3,d_B3,d_C3,nx,ny);
     s2<<<grid,block,0,streams[3]>>>(d_A4,d_B4,d_C4,nx,ny);


     // record stop event
     CHECK(hipEventRecord(stop, 0));
     CHECK(hipEventSynchronize(stop));
 
     // calculate elapsed time
     CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
     printf("Measured time for parallel execution = %f\nstopwatch = %f\n",
            elapsed_time / 1000.0f,s.elapsed());
 
    CHECK(hipMemcpy(h_res1, d_C1, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_res2, d_C2, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_res3, d_C3, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_res4, d_C4, nBytes, hipMemcpyDeviceToHost));
        for(int i=0;i<N;i++)
            {
                if(h_res1[i]!=h_res2[i] || h_res3[i]!=h_res4[i])
                    printf("BAD\n");
            }

    CHECK(hipEventCreateWithFlags(&e_A,hipEventDisableTiming));
    CHECK(hipEventCreateWithFlags(&e_S,hipEventDisableTiming));
    
    // record start event
    s.reset();
    CHECK(hipEventRecord(start, 0));
    
    a1<<<grid,block,0,streams[0]>>>(d_A1,d_B1,d_C1,nx,ny);
    CHECK(hipEventRecord(e_A, 0));
    CHECK(hipStreamWaitEvent(streams[0], e_A, 0));
    a2<<<grid,block,0,streams[1]>>>(d_A2,d_B2,d_C2,nx,ny);
    
    
    s1<<<grid,block,0,streams[2]>>>(d_A3,d_B3,d_C3,nx,ny);
    CHECK(hipEventRecord(e_S, 0));
    CHECK(hipStreamWaitEvent(streams[2], e_S, 0));
    s2<<<grid,block,0,streams[3]>>>(d_A4,d_B4,d_C4,nx,ny);
    

    // record stop event
    CHECK(hipEventRecord(stop, 0));
    CHECK(hipEventSynchronize(stop));

    // calculate elapsed time
    CHECK(hipEventElapsedTime(&elapsed_time, start, stop));
    printf("Measured time for parallel execution with events = %f\nstopwatch = %f\n",
           elapsed_time / 1000.0f,s.elapsed());


    CHECK(hipMemcpy(h_res1, d_C1, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_res2, d_C2, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_res3, d_C3, nBytes, hipMemcpyDeviceToHost));
    CHECK(hipMemcpy(h_res4, d_C4, nBytes, hipMemcpyDeviceToHost));
    for(int i=0;i<N;i++)
    {
        if(h_res1[i]!=h_res2[i] || h_res3[i]!=h_res4[i])
            printf("BAD\n");
    }
     // release all stream
     for (int i = 0 ; i < NSTREAM ; i++)
     {
         CHECK(hipStreamDestroy(streams[i]));
     }
 
     free(streams);
    

    // initialize host array
  
    free(h_A);
    free(h_B);
    free(h_res1);
    free(h_res2);
    free(h_res3);
    free(h_res4);
    CHECK(hipFree(d_A1));CHECK(hipFree(d_A2));CHECK(hipFree(d_A3));CHECK(hipFree(d_A4));
    CHECK(hipFree(d_B1));CHECK(hipFree(d_B2));CHECK(hipFree(d_B3));CHECK(hipFree(d_B4));
    CHECK(hipFree(d_C1));CHECK(hipFree(d_C2));CHECK(hipFree(d_C3));CHECK(hipFree(d_C4));
    CHECK(hipEventDestroy(start));
    CHECK(hipEventDestroy(stop));
    
    CHECK(hipEventDestroy(e_A));
    CHECK(hipEventDestroy(e_S));
    
    CHECK(hipDeviceReset());
    return EXIT_SUCCESS;
}

/* 

a starting main at device 0: GeForce GTX 1050
Measured time for sequence execution = 0.008140
stopwatch = 0.008676
Measured time for parallel execution = 0.008136
stopwatch = 0.008441
Measured time for parallel execution with events = 0.008160
stopwatch = 0.008459

*/