#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

/*
 * This is an example demonstrating usage of the cuSPARSE library to perform a
 * sparse matrix-vector multiplication on randomly generated data.
 */

/*
 * M = # of rows
 * N = # of columns
 */
int M = 5;
int N = 5;

/*
 * Generate a vector of length N with random single-precision floating-point
 * values between 0 and 100.
 */
void generate_random_vector(int N, float **outX)
{
    int i;
    double rMax = (double)RAND_MAX;
    float *X = (float *)malloc(sizeof(float) * N);

    for (i = 0; i < N; i++)
    {
        int r = rand();
        double dr = (double)r;
        X[i] = (dr / rMax) * 100.0;
    }

    *outX = X;
}

void compute_coefficients(int N, int M, float* A, float* y)
{
    for (int i = 0; i < N; i++)
    {
        float s=0.0f;
        for (int j = 0; j < M; j++)
        {
            s=s+A[j * N + i];
        }
        y[i]=s;
    }
}

/*
 * Generate random dense matrix A in column-major order, while rounding some
 * elements down to zero to ensure it is sparse.
 */
int generate_random_dense_matrix(int M, int N, float **outA)
{
    int i, j;
    double rMax = (double)RAND_MAX;
    float *A = (float *)malloc(sizeof(float) * M * N);
    //hipMemset(A,0,M*N*sizeof(float));
    int totalNnz = 0;

    for (j = 0; j < N; j++)
    {
        for (i = 0; i < M; i++)
        {
        
            int r = rand();
            float *curr = A + (j * M + i);

            if (r % 3 > 0)
            {
                *curr = 0.0f;
            }
            else
            {
                double dr = (double)r;
                *curr = (dr / rMax) * 5.0;
            }
            //if(i==j) *curr=1.0f;
            if (*curr != 0.0f)
            {
                totalNnz++;
            }
           
        }
    }

    *outA = A;
    return totalNnz;
}

void print_partial_matrix(float *M, int nrows, int ncols, int max_row,
    int max_col)
{
int row, col;

for (row = 0; row < max_row; row++)
{
    for (col = 0; col < max_col; col++)
    {
        printf("%2.2f ", M[row * ncols + col]);
    }
    printf("...\n");
}
printf("...\n");
}

int main(int argc, char **argv)
{
    int row;
    float *A, *dA;
    int *dNnzPerRow;
    float *dCsrValA;
    int *dCsrRowPtrA;
    int *dCsrColIndA;
    int totalNnz;
    float alpha = 1.0f;
    //float beta = 0.0f;
    float *dX, *X;
    float *dY, *Y;
    int structural_zero;
    int numerical_zero;
    hipsparseHandle_t handle = 0;
    hipsparseMatDescr_t descr = 0;
    csrsv2Info_t  info_A  = 0;
    const hipsparseSolvePolicy_t policy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
    int lworkInBytes;
    void * d_work = NULL;
    // Generate input
    srand(9384);
    int trueNnz = generate_random_dense_matrix(M, N, &A);
    print_partial_matrix(A,M,M,M,M);
    
   
      // generate_random_vector(M, &Y);
    X=(float*)malloc(N*sizeof(float));
    Y=(float*)malloc(M*sizeof(float));
    compute_coefficients(M,N,A,Y);
    for(int i=0;i<N;i++)
        {
            printf("%f ",Y[i]);
            Y[i]=1;
        }
        printf("\n");
    //memset(X,0,N*sizeof(float));
    // Create the cuSPARSE handlef
    CHECK_CUSPARSE(hipsparseCreate(&handle));
    CHECK_CUSPARSE(hipsparseCreateCsrsv2Info(&info_A));
    // Allocate device memory for vectors and the dense form of the matrix A
    CHECK(hipMalloc((void **)&dX, sizeof(float) * N));
    CHECK(hipMalloc((void **)&dY, sizeof(float) * M));
    CHECK(hipMalloc((void **)&dA, sizeof(float) * M * N));
    CHECK(hipMalloc((void **)&dNnzPerRow, sizeof(int) * M));

    // Construct a descriptor of the matrix A
    CHECK_CUSPARSE(hipsparseCreateMatDescr(&descr));
    CHECK_CUSPARSE(hipsparseSetMatFillMode(descr, HIPSPARSE_FILL_MODE_LOWER));
    CHECK_CUSPARSE(hipsparseSetMatDiagType(descr, HIPSPARSE_DIAG_TYPE_NON_UNIT));
    CHECK_CUSPARSE(hipsparseSetMatIndexBase(descr, HIPSPARSE_INDEX_BASE_ZERO));
    CHECK_CUSPARSE(hipsparseSetMatType(descr, HIPSPARSE_MATRIX_TYPE_GENERAL));
    // Transfer the input vectors and dense matrix A to the device
    //CHECK(hipMemcpy(dX, X, sizeof(float) * N, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dY, Y, sizeof(float) * M, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(dA, A, sizeof(float) * M * N, hipMemcpyHostToDevice));
/*
    // Compute the number of non-zero elements in A
    CHECK_CUSPARSE(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, M, N, descr, dA,
                                M, dNnzPerRow, &totalNnz));
*/
    totalNnz=trueNnz;
    if (totalNnz != trueNnz)
    {
        fprintf(stderr, "Difference detected between cuSPARSE NNZ and true "
                "value: expected %d but got %d\n", trueNnz, totalNnz);
        return 1;
    }

    // Allocate device memory to store the sparse CSR representation of A
    CHECK(hipMalloc((void **)&dCsrValA, sizeof(float) * totalNnz));
    CHECK(hipMalloc((void **)&dCsrRowPtrA, sizeof(int) * (M + 1)));
    CHECK(hipMalloc((void **)&dCsrColIndA, sizeof(int) * totalNnz));

    // Convert A from a dense formatting to a CSR formatting, using the GPU
    CHECK_CUSPARSE(hipsparseSdense2csr(handle, M, N, descr, dA, M, dNnzPerRow,
                                      dCsrValA, dCsrRowPtrA, dCsrColIndA));

    for(int i=0;i<N;i++)
        X[i]=1;
                                     
    CHECK(hipMemcpy(dX, X, sizeof(float) * N, hipMemcpyHostToDevice));

    /*
    CHECK_CUSPARSE(hipsparseScsrmv(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                        M, N, totalNnz, &alpha, descr, dCsrValA,
                                        dCsrRowPtrA, dCsrColIndA, dX, &beta, dY));
                             
                                
                                   
    CHECK(hipMemcpy(Y, dY, sizeof(float) * M, hipMemcpyDeviceToHost));
                                  
    for (row = 0; row < 5; row++)
        {
         printf("%f ", Y[row]);
        }
        printf("%\n mnozenje\n");
 */  
        compute_coefficients(M,N,A,Y);
        CHECK(hipMemcpy(dY, Y, sizeof(float) * N, hipMemcpyHostToDevice));

    CHECK_CUSPARSE(hipsparseScsrsv2_bufferSize(
                                handle,
                                HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                M,
                                totalNnz,
                                descr,
                                dCsrValA,
                                dCsrRowPtrA,
                                dCsrColIndA,
                                info_A,
                                &lworkInBytes));

    if (NULL != d_work) { hipFree(d_work); }
    CHECK(hipMalloc((void**)&d_work, lworkInBytes));
    CHECK(hipDeviceSynchronize());
    CHECK_CUSPARSE(hipsparseScsrsv2_analysis(handle,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    M,
                                    totalNnz,
                                    descr,
                                    dCsrValA,
                                    dCsrRowPtrA,
                                    dCsrColIndA,
                                    info_A,
                                    policy,
                                    d_work));
    CHECK(hipDeviceSynchronize());

    hipsparseStatus_t status = hipsparseXcsrsv2_zeroPivot(handle, info_A, &structural_zero);
if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
   printf("L(%d,%d) is missing\n", structural_zero, structural_zero);
}
    CHECK_CUSPARSE(hipsparseScsrsv2_solve(handle,
                                    HIPSPARSE_OPERATION_NON_TRANSPOSE,
                                    M,
                                    totalNnz,
                                    &alpha,
                                    descr,
                                    dCsrValA,
                                    dCsrRowPtrA,
                                    dCsrColIndA,
                                    info_A,
                                    dY,
                                    dX,
                                    policy,
                                    d_work));
    CHECK(hipDeviceSynchronize());
    status = hipsparseXcsrsv2_zeroPivot(handle, info_A, &numerical_zero);
if (HIPSPARSE_STATUS_ZERO_PIVOT == status){
   printf("L(%d,%d) is zero\n", numerical_zero, numerical_zero);
}
    
    // Copy the result vector back to the host
    CHECK(hipMemcpy(Y, dY, sizeof(float) * M, hipMemcpyDeviceToHost));

    for (row = 0; row < 5; row++)
    {
        printf("%2.2f ", Y[row]);
    }

    printf("...\n");

    CHECK(hipMemcpy(X, dX, sizeof(float) * M, hipMemcpyDeviceToHost));

    for (row = 0; row < 5; row++)
    {
        printf("%2.2f ", X[row]);
    }

    printf("...\n\n");

    // Perform matrix-vector multiplication with the CSR-formatted matrix A
  
    
    free(A);
    free(X);
    free(Y);

    CHECK(hipFree(dX));
    CHECK(hipFree(dY));
    CHECK(hipFree(dA));
    CHECK(hipFree(d_work));
    CHECK(hipFree(dNnzPerRow));
    CHECK(hipFree(dCsrValA));
    CHECK(hipFree(dCsrRowPtrA));
    CHECK(hipFree(dCsrColIndA));
    hipsparseDestroyCsrsv2Info(info_A);
    CHECK_CUSPARSE(hipsparseDestroyMatDescr(descr));
    CHECK_CUSPARSE(hipsparseDestroy(handle));


    return 0;
}
/* 
 ovo nece iz ne znam ni ja kog razloga mozda je i neki bug
 a mozda i ja nesto nisam dobro uradio al u sustini po dokumentaciji
 bi trebalo ovako da prodje

*/