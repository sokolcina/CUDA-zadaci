#include "../common/common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

/*
 * A simple example of performing matrix-vector multiplication using the cuBLAS
 * library and some randomly generated inputs.
 */

/*
 * N = # of rows
 * M = # of columns
 */
int N = 500;
int M = 500;



void generate_random_dense_matrix(int N, int M, float *A)
{
    int i, j;
    double rMax = (double)RAND_MAX;

    //int id=1;
    for (i = 0; i < N; i++)
    {
        
        for (j = 0; j < M; j++)
        {
            double dr = (double)rand();
            A[i * M + j] = (dr / rMax) * 100.0; 
            //A[i * M + j]=id++;
        }
    }
}



void compute_coefficients(int N, int M, float* A, float* y)
{
    for (int i = 0; i < N; i++)
    {
        float s=0.0f;
        for (int j = 0; j < M; j++)
        {
            s=s+A[j * N + i];
        }
        y[i]=s;
    }
}

void rearrange(float *vec, int *pivotArray, int N){
    for (int i = 0; i < N; i++) {
        float temp = vec[i];
        vec[i] = vec[pivotArray[i] - 1];
        vec[pivotArray[i] - 1] = temp;
    }   
}

int main(int argc, char **argv)
{
   
    float *h_A, *h_x, *h_y;
    float *d_A, *d_x, *d_y;
    float beta;
    float alpha;
    hipblasHandle_t handle = 0;


    alpha = 1.0f;
    //beta = 0.0f;
    if (argc > 1) {
        N = atoi(argv[1]);
        M = N;
    }
    // Generate inputs
    srand(9384);

    //CHECK(hipDeviceSynchronize());
    h_A = (float*)malloc(N * M * sizeof(float));
    h_y = (float*)malloc(N * sizeof(float));
    h_x = (float*)malloc(M * sizeof(float));

    generate_random_dense_matrix(N, M, h_A);
    compute_coefficients(N, M, h_A, h_y);
    memset(h_x,0,M * sizeof(float));
    CHECK(hipMalloc(&d_y, N * sizeof ( float )));
    CHECK(hipMalloc(&d_x, M * sizeof ( float )));
    CHECK(hipMalloc(&d_A, N * M * sizeof ( float )));
    CHECK_CUBLAS(hipblasSetMatrix(N,M,sizeof(float),h_A,N,d_A,N));
    /*
    for(int i=0; i<M; i++)
        h_x[i]=2.0f; */

     

    hipMemset(d_x, 0, M * sizeof(float));
    
    CHECK(hipMemcpy(d_y, h_y, sizeof(float) * N, hipMemcpyHostToDevice));

    CHECK_CUBLAS(hipblasCreate(&handle));

    //priprema za hipblasSgetrfBatched da dobijemo pivotArray
    //za LU dekompoziciju
    int *P, *Info;
   
    float ** h_AA = (float **) malloc(sizeof(float *));
    *h_AA = d_A;
    float ** d_AA;
    CHECK(hipMalloc(&d_AA,sizeof(float*)));
    CHECK(hipMemcpy(d_AA,h_AA,sizeof(float*),hipMemcpyHostToDevice));

    CHECK(hipMalloc(&P, N * sizeof(int)));
    CHECK(hipMalloc(&Info, sizeof(int)));
    
    
    
    CHECK_CUBLAS(hipblasSgetrfBatched(handle,N,d_AA,N,P,Info,1));
    CHECK(hipDeviceSynchronize());

    int INFOh = 0;
    CHECK(hipMemcpy(&INFOh,Info,sizeof(int),hipMemcpyDeviceToHost));

    printf("%d\n",INFOh);
    if(INFOh == N)
    {
        printf("Factorization Failed: Matrix is singular\n");
        hipDeviceReset();
        exit(EXIT_FAILURE);
    }
    int *h_P=(int*)malloc(N*sizeof(int));
    CHECK(hipMemcpy(h_P,P,N*sizeof(int),hipMemcpyDeviceToHost));
    /*
    for(int i=0;i<N;i++)
    printf("%f ",h_y[i]);
    printf("\n");
   
    for(int i=0;i<N;i++)
    printf("%f ",h_y[i]);
    printf("\n"); */

    //da se preraspodeli
    rearrange(h_y,h_P,N);
    CHECK(hipMemcpy(d_y, h_y, N * sizeof(float), hipMemcpyHostToDevice));
    //CHECK_CUBLAS(hipblasSgetriBatched(handle,N,d_AA,N,P,Info,));
    
    //donji deo matrice
    CHECK_CUBLAS(hipblasStrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, N, 1, &alpha, d_A, N, d_y, N));
    CHECK(hipDeviceSynchronize());
    // gornji deo matrice
    CHECK_CUBLAS(hipblasStrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, N, 1, &alpha, d_A, N, d_y, N));
    CHECK(hipDeviceSynchronize());

    CHECK(hipMemcpy(h_x, d_y, N * sizeof(float), hipMemcpyDeviceToHost));
    
    printf("\n");
    /*for (int i = 0; i < N; i++)
    {
        for (int j = 0; j < M; j++)
        {
            printf("%2.6f ", h_A[j * N + i]);
        }
        printf("\n");
    }*/
    /*for(int i=0;i<N;i++)
        printf("%d ",h_P[i]);
    printf("\n"); */
    printf("X: \n");
    for(int i=0;i<N;i++)
        printf("%2.3f ",h_x[i]);
   
    free(h_A);
    free(h_x);
    free(h_y);
    free(h_AA);
    CHECK(hipFree(d_AA[0]));
    CHECK(hipFree(d_AA));
    CHECK(hipFree(d_x));
    CHECK(hipFree(d_y));
    CHECK(hipFree(Info));
    CHECK(hipFree(P));
  
    CHECK_CUBLAS(hipblasDestroy(handle));

    return 0;
}


/* 

0

X:
1.000 1.000 1.000 1.000 1.001 1.000 1.000 1.000 1.000 1.000 
1.001 0.999 0.999 1.000 0.999 1.001 1.001 1.000 1.000 1.000 
1.001 1.000 1.001 1.000 0.999 0.999 1.001 0.999 0.999 0.998 
1.000 1.001 1.000 0.999 1.000 1.000 1.001 1.000 1.000 1.000 
1.000 1.001 1.000 0.999 1.001 1.001 1.000 0.999 1.000 0.999 
0.999 1.000 0.999 0.999 0.998 1.000 1.000 1.000 1.000 1.000 
0.999 1.000 1.001 0.999 1.000 1.000 1.001 1.000 0.998 1.000 
1.000 1.000 1.001 1.000 1.000 1.000 0.999 1.001 1.000 1.001 
1.000 1.000 1.001 1.000 1.000 0.999 1.000 1.000 1.001 1.001 
1.000 1.001 1.000 1.000 0.999 1.000 1.000 1.001 1.000 1.000 
1.000 1.000 0.999 0.999 0.998 1.000 1.000 1.001 1.001 1.000 
1.000 1.000 0.998 1.000 0.999 1.001 1.000 0.999 1.002 0.999 
1.000 0.998 0.999 1.000 1.000 1.000 1.000 0.999 1.000 0.999 
1.001 0.999 1.001 1.001 1.000 1.000 1.000 1.001 1.001 1.000 
1.000 1.001 1.001 1.001 1.000 1.001 1.001 1.001 1.000 0.999 
1.000 0.999 1.000 1.000 1.001 1.000 1.000 1.000 1.000 0.999 
0.999 1.000 1.000 0.999 1.000 1.000 1.001 1.000 1.000 1.000 
1.000 1.000 0.999 1.002 1.001 0.999 1.000 0.999 1.000 1.000 
1.000 1.000 1.000 1.000 1.000 1.000 1.000 1.000 1.000 0.999 
1.001 0.999 0.999 1.000 1.001 1.000 0.999 1.000 0.998 0.999 
0.999 1.001 1.001 1.001 1.000 1.000 1.000 1.000 1.000 0.999 
1.000 1.001 1.000 1.000 1.000 1.001 0.999 0.999 1.000 0.999 
1.000 1.000 1.002 1.001 1.000 0.999 1.001 1.001 1.001 1.000 
1.000 1.000 1.001 1.000 1.001 1.000 0.999 1.000 0.998 1.000 
1.001 0.999 1.000 0.999 0.998 1.000 0.999 1.000 1.000 1.001 
1.001 0.999 1.000 0.999 1.001 1.000 1.000 1.000 0.999 1.000 
1.000 1.001 1.000 1.002 0.999 1.000 1.000 0.998 0.999 1.000 
1.000 1.000 1.000 1.001 1.000 1.000 1.000 1.000 1.001 1.001 
1.000 1.000 1.001 0.999 1.001 1.001 1.000 1.000 1.000 1.001 
1.001 1.000 1.001 0.999 1.000 1.001 1.000 1.000 0.999 1.000 
1.000 1.000 1.001 0.999 0.999 1.000 1.000 1.000 1.001 1.000 
1.001 0.999 1.001 1.000 0.999 1.000 1.001 1.000 0.999 1.000 
1.001 0.999 1.001 1.000 0.999 1.000 1.000 0.999 1.000 1.000 
1.001 1.000 1.000 1.000 1.000 1.001 0.999 1.001 1.000 0.999 
1.000 1.000 1.000 1.000 1.000 1.000 1.000 1.000 1.001 1.000 
1.000 1.000 1.002 1.000 0.999 1.001 1.001 0.999 1.001 1.000 
1.000 0.999 1.002 1.001 0.998 0.999 1.000 0.999 1.000 0.999 
1.000 1.000 1.001 1.000 1.000 1.001 1.000 1.000 1.000 0.999 
0.999 1.001 0.999 0.999 1.001 1.000 1.000 1.000 1.000 1.000 
1.001 0.998 1.000 1.001 1.001 1.001 1.000 1.000 0.998 0.999 
1.000 0.999 1.000 0.999 1.000 1.000 1.000 1.000 1.001 1.000 
1.000 1.001 0.999 0.998 0.998 1.000 1.001 1.001 0.999 1.001 
0.999 1.000 1.000 1.000 1.001 1.000 1.001 1.001 1.002 1.000 
1.000 1.000 1.000 1.000 1.001 1.000 1.000 1.001 1.001 1.001 
1.000 0.999 1.001 1.000 1.001 1.000 1.000 1.000 1.002 1.001 
1.000 0.999 1.000 1.001 1.000 0.999 1.001 0.998 1.002 1.000 
1.001 0.999 1.000 1.001 1.000 1.000 1.000 1.000 0.999 1.001 
0.999 1.000 1.001 1.000 1.001 1.000 0.999 0.999 1.000 1.000 
1.000 1.001 1.000 0.999 1.000 1.001 0.999 1.001 1.000 1.000 
0.999 0.999 1.000 1.002 1.000 1.000 1.000 1.000 0.999 1.000 

*/